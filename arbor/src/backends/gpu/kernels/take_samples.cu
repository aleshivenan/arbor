#include "hip/hip_runtime.h"
#include <common_types.hpp>
#include <backends/event.hpp>
#include <backends/fvm_types.hpp>
#include <backends/multi_event_stream_state.hpp>

namespace arb {
namespace gpu {

namespace kernels {
    __global__ void take_samples(
	    multi_event_stream_state<raw_probe_info> s,
	    const fvm_value_type* time,
	    fvm_value_type* sample_time,
	    fvm_value_type* sample_value)
    {
        int i = threadIdx.x+blockIdx.x*blockDim.x;

        if (i<s.n) {
            auto begin = s.ev_data+s.begin_offset[i];
            auto end = s.ev_data+s.end_offset[i];
            for (auto p = begin; p!=end; ++p) {
                sample_time[p->offset] = time[i];
                sample_value[p->offset] = *p->handle;
            }
        }
    }
}

void take_samples(
	const multi_event_stream_state<raw_probe_info>& s,
	const fvm_value_type* time,
	fvm_value_type* sample_time,
	fvm_value_type* sample_value)
{
    if (!s.n_streams()) {
        return;
    }

    constexpr int blockwidth = 128;
    int nblock = 1+(s.n_streams()-1)/blockwidth;
    kernels::take_samples<<<nblock, blockwidth>>>(s, time, sample_time, sample_value);
}

} // namespace gpu
} // namespace arb

