#include "hip/hip_runtime.h"
#include <cstdint>

#include <constants.hpp>

#include "../nernst.hpp"
#include "detail.hpp"

namespace arb {
namespace gpu {

namespace kernels {
    template <typename T>
    __global__ void nernst(std::size_t n, int valency, T temperature, const T* Xo, const T* Xi, T* eX) {
        auto i = threadIdx.x+blockIdx.x*blockDim.x;

        // factor 1e3 to scale from V -> mV
        constexpr T RF = 1e3*constant::gas_constant/constant::faraday;
        T factor = RF*temperature/valency;
        if (i<n) {
            eX[i] = factor*std::log(Xo[i]/Xi[i]);
        }
    }
} // namespace kernels

void nernst(std::size_t n,
            int valency,
            fvm_value_type temperature,
            const fvm_value_type* Xo,
            const fvm_value_type* Xi,
            fvm_value_type* eX)
{
    constexpr int block_dim = 128;
    const int grid_dim = impl::block_count(n, block_dim);
    kernels::nernst<<<grid_dim, block_dim>>>
        (n, valency, temperature, Xo, Xi, eX);
}

} // namespace gpu
} // namespace arb
